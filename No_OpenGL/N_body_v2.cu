#include "hip/hip_runtime.h"
/* second version of N body simulation using CUDA */

#include <iostream>
#include <fstream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <ctime>

using namespace std;


/* define the global constants */
const double G = 6.67 * pow(10, -11);
const double e = 0.00001;
const double period = 1;


/* define the structure of particle */
struct particle
{
	double m;
	double pos_x;
	double pos_y;
	double v_x;
	double v_y;
	double a_x;
	double a_y;

	particle(double m = 0, double pos_x = 0, double pos_y = 0, 
			double v_x = 0, double v_y = 0, double a_x = 0, double a_y = 0)
	{
		this->m			= m;
		this->pos_x		= pos_x;
		this->pos_y		= pos_y;
		this->v_x		= v_x;
		this->v_y		= v_y;
		this->a_x		= a_x;
		this->a_y		= a_y;
	}
};


struct my_double2
{
	double x, y;
	
	__device__ my_double2(double x = 0, double y = 0)
	{
		this->x = x;
		this->y = y;
	}
};



/* define the global data */
int g_N;									// number of particles
int g_P;									// number of particles in a tile
thrust::host_vector<particle> g_pv;			// particle vector


void setUp();



/* calculate the interaction between two bodies */
__device__ my_double2 bodyBodyAcceleration(double G, double e, particle b1, particle b2, my_double2 acceleration)
{	
	double r_2 = pow((b1.pos_x - b2.pos_x),2) + pow((b1.pos_y - b2.pos_y),2);
	b1.a_x = (-1) * G * b2.m * (b1.pos_x - b2.pos_x) / (pow(r_2 + e, 1.5));
	b1.a_y = (-1) * G * b2.m * (b1.pos_y - b2.pos_y) / (pow(r_2 + e, 1.5));
	
	acceleration.x += b1.a_x;
	acceleration.y += b1.a_y;
	return acceleration;
}


/* calculate the interaction inside a P*P block */
__device__ my_double2 tileAcceleration(double G, double e, particle b, my_double2 acceleration)
{
	extern __shared__ particle shParticles[];
	
	
	for(int i = 0; i < blockDim.x; ++i)
	{
		acceleration = bodyBodyAcceleration(G, e, b, shParticles[i], acceleration);
	}
	
	return acceleration;
}


/* update the position */
__device__ void updatePosition(double period, particle* particle_arr)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	/* compute the velocity */
	particle_arr[idx].v_x += particle_arr[idx].a_x * period;
	particle_arr[idx].v_y += particle_arr[idx].a_y * period;
	
	/* compute the new position */

	particle_arr[idx].pos_x += particle_arr[idx].v_x * period;
	particle_arr[idx].pos_y += particle_arr[idx].v_y * period;
}


/* calculate the whole acceleration */
__global__ void updateScene(int N, int P, double G, double e, double period, particle* particle_arr)
{
	extern __shared__ particle shParticles[];
	
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	particle ptc = particle_arr[id];
	
	my_double2 acceleration;
	acceleration.x = ptc.a_x;
	acceleration.y = ptc.a_y;
	
	int i, tile;
	for(i = 0, tile = 0; i < N; i += P, ++tile)
	{
		/* fill in the shared memory */
		int idx = tile * blockDim.x + threadIdx.x;
		shParticles[threadIdx.x] = particle_arr[idx];
		__syncthreads();
		
		/* calculate the acceleration with a tile */
		acceleration = tileAcceleration(G, e, ptc, acceleration);
		__syncthreads();	
	}
	
	ptc.a_x = acceleration.x;
	ptc.a_y = acceleration.y;
	
	updatePosition(period, particle_arr);
}



int main(int argc, char ** argv) {

	setUp();
	g_P = static_cast<int>(sqrt(g_N)) + 1;
	
	/* device copy of particle array */
	thrust::device_vector<particle> d_particle_arr = g_pv;
	
	/* get the raw pointer of particle array */
	particle *particle_arr = thrust::raw_pointer_cast(d_particle_arr.data());
	
	clock_t start, finish;
	start = clock();
	
	int time = 0;
	while(time < 100000)
	{	
		
		updateScene<<<g_P,g_P,g_P*sizeof(particle)>>>(g_N, g_P, G, e, period, particle_arr);
		/*
		g_pv = d_particle_arr;
		
		
		for ( int i = 0; i < g_N; ++i )
		{
			cout << "particle: " << i << " pos_x: " << g_pv[i].pos_x << " pos_y: " << g_pv[i].pos_y << endl;
		}
		*/
		time++;
	}
	
	
	finish = clock();
	cout << "Execution Time: " << (double)(finish-start)/CLOCKS_PER_SEC << endl;
	
	return 0;
}



/* read the input data */
void setUp()
{
	ifstream inFile;
	inFile.open("input.txt");
	
	inFile >> g_N;
	g_pv.resize(g_N);
	for ( int i = 0; i < g_N; ++i )
	{
		inFile >> g_pv[i].m >> g_pv[i].pos_x >> g_pv[i].pos_y
			   >> g_pv[i].v_x >> g_pv[i].v_y >> g_pv[i].a_x >> g_pv[i].a_y; 
	}
	
	inFile.close();
}

